#include "bits/stdc++.h"
#include <hip/hip_runtime.h>
#define in std::cin
#define out std::cout
#define rep(i,N) for(LL i=0;i<N;++i)
typedef long long int LL;

#define CHECK(call)																			\
{																							\
	const hipError_t error = call;															\
	if (error != hipSuccess)																\
	{																						\
		out << "Error: " << __FILE__ << ":" << __LINE__ << ", ";							\
		out << "code:" << error << ", reason: " << hipGetErrorString(error) << std::endl;	\
		exit(1);																			\
	}																						\
}																							\

__global__ void calcSum(int n, int *a, int& s)
{
	rep(i, n) s += a[i];
}

int main()
{
	int N;
	in >> N;
	size_t nBytes = sizeof(int) * N;

	int *h_A, *d_A, *h_S, *d_S;
	h_A = (int *)malloc(nBytes);
	CHECK(hipMalloc((int**)&d_A, nBytes));
	h_S = (int *)malloc(sizeof(int));
	CHECK(hipMalloc(&d_S, sizeof(int)));

	rep(i, N) h_A[i] = i + 1;

	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	calcSum<<< 1, 1 >>>(N, d_A, *d_S);
	CHECK(hipDeviceSynchronize());
	CHECK(hipMemcpy(h_S, d_S, sizeof(int), hipMemcpyDeviceToHost));
	out << *h_S << std::endl;
	
	free(h_A);
	CHECK(hipFree(d_A));
	free(h_S);
	CHECK(hipFree(d_S));
	CHECK(hipDeviceReset());
}
