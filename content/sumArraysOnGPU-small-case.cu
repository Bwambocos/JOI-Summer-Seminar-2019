#include "bits/stdc++.h"
#include <hip/hip_runtime.h>
#define in std::cin
#define out std::cout
#define rep(i,N) for(LL i=0;i<N;++i)
typedef long long int LL;

// #define CHECK(call)
// {
// 	const cudaError_t error = call;
// 	if (error != cudaSuccess)
// 	{
// 		out << "Error: " << __FILE__ << ":" << __LINE__ << ", ";
// 		out << "code:" << error << ", reason: " << cudaGetErrorString(error) << std::endl;
// 		exit(1);
// 	}
// }

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	bool match = 1;

	rep(i, N)
	{
		if (abs(hostRef[i] - gpuRef[i]) > epsilon)
		{
			match = 0;
			out << "Arrays do not match!" << std::endl;
			out << "host " << hostRef[i] << " gpu " << gpuRef[i] << " at current " << i << std::endl;
			break;
		}
	}

	if (match) out << "Arrays match.\n\n";
}

void initialData(float* ip, int size)
{
	time_t t;
	srand((unsigned int) time(&t));
	rep(i, size) ip[i] = (float)(rand() & 0xFF) / 10.0f;
}

void sumArraysOnHost(float* A, float* B, float* C, const int N)
{
	rep(i, N) C[i] = A[i] + B[i];
}

__global__ void sumArraysOnGPU(float* A, float* B, float* C)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
	out << argv[0] << " Starting...\n";

	int dev = 0;
	hipSetDevice(dev);

	int nElem = 32;
	out << "Vector size " << nElem << "\n";

	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	initialData(h_A, nElem);
	initialData(h_B, nElem);

	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);

	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice);

	dim3 block(1);
	dim3 grid(nElem);

	sumArraysOnGPU<<< grid, block >>>(d_A, d_B, d_C);
	out << "Execution configure <<<" << grid.x << ", " << block.x << ">>>\n";

	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	sumArraysOnHost(h_A, h_B, hostRef, nElem);

	checkResult(hostRef, gpuRef, nElem);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	hipDeviceReset();
}
