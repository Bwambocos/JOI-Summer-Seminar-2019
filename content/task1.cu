#include "bits/stdc++.h"
#include <hip/hip_runtime.h>
#define in std::cin
#define out std::cout
#define rep(i,N) for(LL i=0;i<N;++i)
typedef long long int LL;

#define CHECK(call)	\
{	\
	const hipError_t error = call;	\
	if (error != hipSuccess)	\
	{	\
		out << "Error: " << __FILE__ << ":" << __LINE__ << ", ";	\
		out << "code:" << error << ", reason: " << hipGetErrorString(error) << std::endl;	\
		exit(1);	\
	}	\
}	\

__global__ void addOne(int& n)
{
	n++;
}

int main()
{
	size_t nBytes = sizeof(int);

	int *h_A;
	h_A = (int *)malloc(nBytes);

	int *d_A;
	CHECK(hipMalloc(&d_A, nBytes));

	addOne<<< 100, 100 >>>(*d_A);
	CHECK(hipDeviceSynchronize());

	CHECK(hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost));
	out << *h_A << std::endl;
	
	free(h_A);
	CHECK(hipFree(d_A));
	CHECK(hipDeviceReset());
}
