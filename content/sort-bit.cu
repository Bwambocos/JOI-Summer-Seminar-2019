#include "bits/stdc++.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define in std::cin
#define out std::cout
#define rep(i,N) for(LL i=0;i<N;++i)
typedef long long int LL;

double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

const LL inf = RAND_MAX;

__global__ void solveOnGPU(LL A[], const LL block, const LL step)
{
	LL idx = blockIdx.x * blockDim.x + threadIdx.x;
	LL e = (idx ^ step);
	if (e > idx)
	{
		LL v1 = A[idx];
		LL v2 = A[e];
		if (((idx & block) != 0 && v1 < v2)
			|| ((idx & block) == 0) && v1 > v2)
		{
			A[e] = v1;
			A[idx] = v2;
		}
	}
}

int main()
{
	LL N;
	in >> N;

	// �u���b�N�E�X���b�h���̏�����擾
	int dev = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	LL maxThreads = deviceProp.maxThreadsPerBlock;

	// ���������m��
	LL newN = 1;
	while (newN < N) newN *= 2;
	size_t nBytes = sizeof(LL) * N, newNBytes = sizeof(LL) * newN;
	LL *input, *d_Array, *output;
	input = (LL *)malloc(nBytes);
	output = (LL *)malloc(nBytes);
	hipMalloc((LL**)&d_Array, newNBytes);

	// �����������
	srand((unsigned int)time(0));
	rep(i, N) input[i] = rand();
	
	// GPU �������Ԃ��v��
	auto gpuStartTime = cpuSecond();
	
	LL *temp;
	temp = (LL *)malloc(newNBytes);
	memcpy(temp, input, nBytes);
	for (LL i = N; i < newN; ++i) temp[i] = inf;
	hipMemcpy(d_Array, temp, newNBytes, hipMemcpyHostToDevice);
	for (LL block = 2; block <= newN; block *= 2)
	{
		for (LL step = block / 2; step >= 1; step /= 2)
		{
			solveOnGPU<<< std::max(newN / maxThreads, 1LL), std::min(newN, maxThreads) >>>(d_Array, block, step);
		}
	}
	hipMemcpy(temp, d_Array, newNBytes, hipMemcpyDeviceToHost);
	memcpy(output, temp, nBytes);
	free(temp);

	auto gpuEndTime = cpuSecond();

	out << (gpuEndTime - gpuStartTime) * 1000 << "ms" << std::endl;

	// ��n��
	free(input);
	free(output);
	hipFree(d_Array);
}
